#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/extrema.h>

#include "BoundingBox.h"
#include "Const.h"

struct point3d
{
	float x, y, z;

	__host__ __device__
		point3d() : x(0), y(0), z(0) {}

	__host__ __device__
		point3d(float _x, float _y, float _z) : x(_x), y(_y), z(_z) {}
};

struct bbox
{
	point3d lower_left, upper_right;

	// --- Empty box constructor
	__host__ __device__ bbox() {}

	// --- Construct a box from a single point
	__host__ __device__ bbox(const point3d &point) 
		: lower_left(point), upper_right(point) {}

	// construct a box from a single point
	__host__ __device__	bbox& operator=(const point3d &point)
	{
		lower_left = point;
		upper_right = point;
		return *this;
	}

	// --- Construct a box from a pair of points
	__host__ __device__ bbox(const point3d &ll, const point3d &ur)
		: lower_left(ll), upper_right(ur) {}
};

// --- Reduce a pair of bounding boxes (a, b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox, bbox, bbox>
{
	__host__ __device__ bbox operator()(bbox a, bbox b)
	{
		// --- Lower left corner
		point3d ll(thrust::min(a.lower_left.x, b.lower_left.x),
			thrust::min(a.lower_left.y, b.lower_left.y), 
			thrust::min(a.lower_left.z, b.lower_left.z));

		// --- Upper right corner
		point3d ur(thrust::max(a.upper_right.x, b.upper_right.x),
			thrust::max(a.upper_right.y, b.upper_right.y), 
			thrust::max(a.upper_right.z, b.upper_right.z));

		return bbox(ll, ur);
	}
};

void BoundingBoxGPU(const float *v, const size_t szV, float &xmin, float &xmax,
	float &ymin, float &ymax, float &zmin, float &zmax)
{
	const size_t N = szV / BYTES_PER_VERTEX;
	//thrust::device_vector<point3d> points(N);
	thrust::device_ptr<point3d> points((point3d*)v);

	// --- The initial bounding box contains the first point of the point cloud
	bbox init = bbox(points[0], points[0]);
	
	// --- Binary reduction operation
	bbox_reduction binary_op;

	// --- Compute the bounding box for the point set
	bbox result = thrust::reduce(points, points + N, init, binary_op);

	xmin = result.lower_left.x;
	xmax = result.upper_right.x;
	ymin = result.lower_left.y;
	ymax = result.upper_right.y;
	zmin = result.lower_left.z;
	zmax = result.upper_right.z;
}